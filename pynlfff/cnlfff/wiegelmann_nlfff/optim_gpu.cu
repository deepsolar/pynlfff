/*
 * @Description: https://doi.org/10.1007/s11207-012-9966-z
 * @Author: Thomas Wiegelmann (wiegelmann@mps.mpg.de)
 * @Date: 
 * @LastEditors: Xinze Zhang (zhangxinze17@mails.ucas.ac.cn)
 * @LastEditDescription: Update GPU version
 * @LastEditTime: 202212
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define MCENTERGRAD(f, id) ((f[i + id] - f[i - id]) / doubled_h)
#define MLEFTGRAD(f, id) ((-3 * f[i] + 4 * f[i + id] - f[i + (id << 1)]) / doubled_h)
#define MRIGHTGRAD(f, id) ((+3 * f[i] - 4 * f[i - id] + f[i - (id << 1)]) / doubled_h)

/* GRAD for Boundaries , all 3 differences needed */
#define GRADX(f, i) ((ix > 0 && ix < nx - 1) ? (MCENTERGRAD(f, nynz)) : ((ix == 0) ? (MLEFTGRAD(f, nynz)) : ((ix == nx - 1) ? (MRIGHTGRAD(f, nynz)) : (0.0))))
#define GRADY(f, i) ((iy > 0 && iy < ny - 1) ? (MCENTERGRAD(f, nz)) : ((iy == 0) ? (MLEFTGRAD(f, nz)) : ((iy == ny - 1) ? (MRIGHTGRAD(f, nz)) : (0.0))))
#define GRADZ(f, i) ((iz > 0 && iz < nz - 1) ? (MCENTERGRAD(f, 1)) : ((iz == 0) ? (MLEFTGRAD(f, 1)) : ((iz == nz - 1) ? (MRIGHTGRAD(f, 1)) : (0.0))))

#define GRADX_INNER(f, i) (MCENTERGRAD(f, nynz))
#define GRADY_INNER(f, i) (MCENTERGRAD(f, nz))
#define GRADZ_INNER(f, i) (MCENTERGRAD(f, 1))

extern double time1_calculate, time2_calculate, timeTot_calculate;
extern double time1_calculate1, time2_calculate1, timeTot_calculate1;
extern double time1_calculate2, time2_calculate2, timeTot_calculate2;
extern double time1_calculate3, time2_calculate3, timeTot_calculate3;
extern double time1_calculate4, time2_calculate4, timeTot_calculate4;

__global__ void green_d(const int nx, const int ny, const int nz, const int cube_x, const int cube_y, const int cube_z, const double *x, const double *y, const double *z, const double *bz0, double *Pot0)
{
    int i, i2, ix, iy, iz, ix1, iy1, nynz = ny * nz;
    double dummy1, r, rx, ry, rz;

    int delta_x = threadIdx.x * cube_x;
    int delta_y = threadIdx.y * cube_y;
    int delta_z = blockIdx.z * cube_z;
    int max_x = (delta_x + cube_x);
    int max_y = (delta_y + cube_y);
    int max_z = (delta_z + cube_z);
    max_x = max_x < nx ? max_x : nx;
    max_y = max_y < ny ? max_y : ny;
    max_z = max_z < nz ? max_z : nz;

    for (ix = delta_x; ix < max_x; ix++)
        for (iy = delta_y; iy < max_y; iy++)
            for (iz = delta_z; iz < max_z; iz++)
            {
                i = ix * nynz + iy * nz + iz;
                dummy1 = 0.0;
                for (ix1 = 0; ix1 < nx; ix1++)
                    for (iy1 = 0; iy1 < ny; iy1++)
                    {
                        i2 = ny * ix1 + iy1;
                        rx = x[ix] - x[ix1];
                        ry = y[iy] - y[iy1];
                        rz = z[iz];
                        r = sqrt(rx * rx + ry * ry + rz * rz);
                        dummy1 = dummy1 - bz0[i2] / r;
                    }
                Pot0[i] = dummy1 / 2.0 / 3.14159;
            }
}

void green_(double *Bx, double *By, double *Bz)
{
    FILE *streamw, *initfile;
    double zoff, dummy1, doubled_h;
    int nx, ny, nz, nynz, nxnynz;
    int i, i2, ix, iy, iz;
    doubled_h = 1.0 * 2;
    char leer[25];

    if ((initfile = fopen("grid.ini", "r")) == NULL)
    {
        printf("\n Error grid.ini");
        exit(1);
    }
    fscanf(initfile, "%s %i", &leer, &nx);
    fscanf(initfile, "%s %i", &leer, &ny);
    fscanf(initfile, "%s %i", &leer, &nz);
    fscanf(initfile, "%s %lf", &leer, &zoff);
    fclose(initfile);
    printf("\n nx= %i, ny=%i , nz= %i \n", nx, ny, nz);
    nynz = ny * nz;
    nxnynz = nx * ny * nz;
    zoff = 0.5;

    double *x, *y, *z, *bz0, *Pot0;
    x = (double *)calloc(nx, sizeof(double));
    y = (double *)calloc(ny, sizeof(double));
    z = (double *)calloc(nz, sizeof(double));
    bz0 = (double *)calloc(nx * ny, sizeof(double));
    Pot0 = (double *)calloc(nxnynz, sizeof(double));

    double *x_d, *y_d, *z_d, *bz0_d, *Pot0_d;
    hipMalloc((void **)&x_d, nx * sizeof(double));
    hipMalloc((void **)&y_d, ny * sizeof(double));
    hipMalloc((void **)&z_d, nz * sizeof(double));
    hipMalloc((void **)&bz0_d, nx * ny * sizeof(double));
    hipMalloc((void **)&Pot0_d, nxnynz * sizeof(double));

    if ((streamw = fopen("allboundaries.dat", "r")) == NULL)
    {
        printf("\n Error ");
        exit(1);
    }
    for (iy = 0; iy < ny; iy++)
        for (ix = 0; ix < nx; ix++)
        {
            i2 = ny * ix + iy;
            fscanf(streamw, "%lf", &dummy1);
            fscanf(streamw, "%lf", &dummy1);
            fscanf(streamw, "%lf", &dummy1);
            bz0[i2] = dummy1;
        }
    fclose(streamw);
    printf("\n Vectormagnetogram loaded");
    printf("\n Only Bz is used for potential field \n");

    /***********************************************************/
    for (ix = 0; ix < nx; ix++)
    {
        x[ix] = ix * 1.0;
    }
    for (iy = 0; iy < ny; iy++)
    {
        y[iy] = iy * 1.0;
    }
    for (iz = 0; iz < nz; iz++)
    {
        z[iz] = zoff + iz * 1.0;
    }

    /* Calculate Potential */

    int cube_x, cube_y, cube_z, ncube_x, ncube_y, ncube_z;
    cube_x = (nx + 31) / 32;
    cube_y = (ny + 31) / 32;
    cube_z = (nz + 7) / 8;
    cube_x = cube_x < 4 ? 4 : cube_x;
    cube_y = cube_y < 4 ? 4 : cube_y;
    cube_z = cube_z < 4 ? 4 : cube_z;
    ncube_x = (nx + cube_x - 1) / cube_x;
    ncube_y = (ny + cube_y - 1) / cube_y;
    ncube_z = (nz + cube_z - 1) / cube_z;

    dim3 blockSize(ncube_x, ncube_y, 1);
    dim3 gridSize(1, 1, ncube_z);

    hipMemcpy((void *)x_d, (void *)x, nx * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)y_d, (void *)y, ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)z_d, (void *)z, nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)bz0_d, (void *)bz0, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Pot0_d, (void *)Pot0, nxnynz * sizeof(double), hipMemcpyHostToDevice);

    green_d<<<gridSize, blockSize>>>(nx, ny, nz, cube_x, cube_y, cube_z, x_d, y_d, z_d, bz0_d, Pot0_d);

    hipMemcpy((void *)Pot0, (void *)Pot0_d, nxnynz * sizeof(double), hipMemcpyDeviceToHost);

    /* Write to Binary */
    for (ix = 0; ix < nx; ix++)
        for (iy = 0; iy < ny; iy++)
            for (iz = 0; iz < nz; iz++)
            {
                i = ix * nynz + iy * nz + iz;
                Bx[i] = GRADX(Pot0, i);
                By[i] = GRADY(Pot0, i);
                Bz[i] = GRADZ(Pot0, i);
            }

    if ((streamw = fopen("B0.bin", "wb")) == NULL)
    {
        printf("\n Error B0.bin");
        exit(1);
    }
    fwrite(Bx, sizeof(double) * nxnynz, 1, streamw);
    fwrite(By, sizeof(double) * nxnynz, 1, streamw);
    fwrite(Bz, sizeof(double) * nxnynz, 1, streamw);
    fclose(streamw);
    printf("\n\n B written to B0.bin \n");
}

void green(double *Bx, double *By, double *Bz)
{
    FILE *streamw, *initfile;
    double zoff, dummy1, doubled_h, r, rx, ry, rz;
    int nx, ny, nz, nynz, nxnynz;
    int i, i2, /*i3,*/ ix, iy, iz, ix1, iy1;
    doubled_h = 1.0 * 2;
    char leer[25];

    if ((initfile = fopen("grid.ini", "r")) == NULL)
    {
        printf("\n Error grid.ini");
        exit(1);
    }
    fscanf(initfile, "%s %i", &leer, &nx);
    fscanf(initfile, "%s %i", &leer, &ny);
    fscanf(initfile, "%s %i", &leer, &nz);
    fscanf(initfile, "%s %lf", &leer, &zoff);
    fclose(initfile);
    printf("\n nx= %i, ny=%i , nz= %i \n", nx, ny, nz);
    nynz = ny * nz;
    nxnynz = nx * ny * nz;
    zoff = 0.5;

    double *x, *y, *z, *bz0, *Pot0;
    x = (double *)calloc(nx, sizeof(double));
    y = (double *)calloc(ny, sizeof(double));
    z = (double *)calloc(nz, sizeof(double));
    Pot0 = (double *)calloc(nxnynz, sizeof(double));
    bz0 = (double *)calloc(nx * ny, sizeof(double));

    if ((streamw = fopen("allboundaries.dat", "r")) == NULL)
    {
        printf("\n Error ");
        exit(1);
    }
    for (iy = 0; iy < ny; iy++)
        for (ix = 0; ix < nx; ix++)
        {
            i2 = ny * ix + iy;
            fscanf(streamw, "%lf", &dummy1);
            fscanf(streamw, "%lf", &dummy1);
            fscanf(streamw, "%lf", &dummy1);
            bz0[i2] = dummy1;
        }
    fclose(streamw);
    printf("\n Vectormagnetogram loaded");
    printf("\n Only Bz is used for potential field \n");

    /***********************************************************/
    for (ix = 0; ix < nx; ix++)
    {
        x[ix] = ix * 1.0;
    }
    for (iy = 0; iy < ny; iy++)
    {
        y[iy] = iy * 1.0;
    }
    for (iz = 0; iz < nz; iz++)
    {
        z[iz] = zoff + iz * 1.0;
    }

/* Calculate Potential */
#ifdef _OPENMP
#pragma omp parallel for private(i, ix, iy, iz, dummy1, ix1, iy1, rx, ry, rz, r, i2)
#endif
    for (ix = 0; ix < nx; ix++)
    {
        printf("\n percent finished = %lf", 100.0 * ix / nx);
        for (iy = 0; iy < ny; iy++)
            for (iz = 0; iz < nz; iz++)
            {
                i = ix * nynz + iy * nz + iz;
                dummy1 = 0.0;
                for (ix1 = 0; ix1 < nx; ix1++)
                    for (iy1 = 0; iy1 < ny; iy1++)
                    {
                        i2 = ny * ix1 + iy1;
                        rx = x[ix] - x[ix1];
                        ry = y[iy] - y[iy1];
                        rz = z[iz];
                        r = sqrt(rx * rx + ry * ry + rz * rz);
                        dummy1 = dummy1 - bz0[i2] / r;
                    }
                Pot0[i] = dummy1 / 2.0 / 3.14159;
            }
    }
/* Write to Binary */
#ifdef _OPENMP
#pragma omp parallel for private(i, ix, iy, iz)
#endif
    for (ix = 0; ix < nx; ix++)
        for (iy = 0; iy < ny; iy++)
            for (iz = 0; iz < nz; iz++)
            {
                i = ix * nynz + iy * nz + iz;
                Bx[i] = GRADX(Pot0, i);
                By[i] = GRADY(Pot0, i);
                Bz[i] = GRADZ(Pot0, i);
            }

    if ((streamw = fopen("B0.bin", "wb")) == NULL)
    {
        printf("\n Error B0.bin");
        exit(1);
    }
    fwrite(Bx, sizeof(double) * nxnynz, 1, streamw);
    fwrite(By, sizeof(double) * nxnynz, 1, streamw);
    fwrite(Bz, sizeof(double) * nxnynz, 1, streamw);
    fclose(streamw);
    printf("\n\n B written to B0.bin \n");
}

__global__ void sum1D_d(int nx, int slice_nx, double *x)
{
    int ix;
    int delta_x = threadIdx.x * slice_nx;
    int max_x = (delta_x + slice_nx);
    max_x = max_x < nx ? max_x : nx;

    for (ix = delta_x + 1; ix < max_x; ix++)
    {
        x[delta_x] += x[ix];
    }

    __syncthreads();
    if (threadIdx.x == 0)
    {
        for (ix = slice_nx; ix < nx; ix += slice_nx)
            x[0] += x[ix];
    }
}

// this func will change x_d
double sum1D(int nx, double *x_d)
{
    int slice_nx = (nx + 63) / 64;
    int nslice_x = (nx + slice_nx - 1) / slice_nx;

    dim3 blockSize(nslice_x, 1, 1);
    dim3 gridSize(1);
    double sum;

    sum1D_d<<<gridSize, blockSize>>>(nx, slice_nx, x_d);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("[sum1D_d]: CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy((void *)&sum, (void *)x_d, 1 * sizeof(double), hipMemcpyDeviceToHost);

    return sum;
}

__global__ void calculateL3_d(const int nx, const int ny, const int nz, const int rect_ny, double *help, const double *mask, const double *Bx, const double *By, const double *Bz, const double *Bxorig, const double *Byorig, const double *Bzorig)
{
    int nynz = ny * nz, i, i1, ix, iy = threadIdx.y;

    int delta_y = threadIdx.y * rect_ny;
    int max_y = (delta_y + rect_ny);
    max_y = max_y < ny ? max_y : ny;

    for (iy = delta_y; iy < max_y; iy++)
    {
        help[iy] = 0.0;
        for (ix = 0; ix < nx; ix++)
        {
            i = ix * nynz + iy * nz;
            i1 = ix * ny + iy;
            help[iy] = help[iy] + mask[i1] * ((Bx[i] - Bxorig[i1]) * (Bx[i] - Bxorig[i1]) + (By[i] - Byorig[i1]) * (By[i] - Byorig[i1]) + (Bz[i] - Bzorig[i1]) * (Bz[i] - Bzorig[i1]));
        }
    }

    __syncthreads();
    if (threadIdx.y == 0)
    {
        for (iy = 1; iy < ny; iy++)
            help[0] += help[iy];
    }
}

double calculateL3(int nx, int ny, int nz, double nave, double *help_d, const double *mask_d, const double *Bx_d, const double *By_d, const double *Bz_d, const double *Bxorig_d, const double *Byorig_d, const double *Bzorig_d)
{
    int rect_ny = (ny + 1023) / 1024;
    int nrect_y = (ny + rect_ny - 1) / rect_ny;

    dim3 blockSize(1, nrect_y, 1);
    dim3 gridSize(1);
    double L3 = 0.0;

    calculateL3_d<<<gridSize, blockSize>>>(nx, ny, nz, rect_ny, help_d, mask_d, Bx_d, By_d, Bz_d, Bxorig_d, Byorig_d, Bzorig_d);
    hipMemcpy((void *)&L3, (void *)help_d, 1 * sizeof(double), hipMemcpyDeviceToHost);
    L3 = L3 / nave;

    return L3;
}

__global__ void calculateL_d(const int *shape, const double doubled_h,
                             const double *Bx, const double *By, const double *Bz,
                             float *DivB, float *odotb, float *oxbx, float *oxby, float *oxbz,
                             float *oxjx, float *oxjy, float *oxjz, float *oxa, float *oya, float *oza, float *oxb, float *oyb, float *ozb,
                             double *helpLs)
{
    int nx = shape[0], ny = shape[1], nz = shape[2], cube_x = shape[3], cube_y = shape[4], cube_z = shape[5], ncube_x = shape[6], ncube_y = shape[7], ncube_z = shape[8];
    double bx, by, bz, cbx, cby, cbz, fx, fy, fz;
    double divB, b2;
    double helpL = 0.0, helpL1 = 0.0, helpL2 = 0.0;
    double o2a, o2b;
    int ix, iy, iz, i;
    int nynz = ny * nz;
    int icube = blockIdx.z * ncube_y * ncube_x + threadIdx.y * ncube_x + threadIdx.x;
    int ncube_xyz = ncube_z * ncube_y * ncube_x;
    int delta_x = threadIdx.x * cube_x;
    int delta_y = threadIdx.y * cube_y;
    int delta_z = blockIdx.z * cube_z;
    int max_x = (delta_x + cube_x);
    int max_y = (delta_y + cube_y);
    int max_z = (delta_z + cube_z);
    max_x = max_x < nx ? max_x : nx;
    max_y = max_y < ny ? max_y : ny;
    max_z = max_z < nz ? max_z : nz;

    for (ix = delta_x; ix < max_x; ix++)
        for (iy = delta_y; iy < max_y; iy++)
            for (iz = delta_z; iz < max_z; iz++)
            {
                i = ix * nynz + iy * nz + iz;
                bx = Bx[i];
                by = By[i];
                bz = Bz[i];

                b2 = (bx * bx + by * by + bz * bz);

                cbx = GRADY(Bz, i) - GRADZ(By, i);
                cby = GRADZ(Bx, i) - GRADX(Bz, i);
                cbz = GRADX(By, i) - GRADY(Bx, i);
                divB = GRADX(Bx, i) + GRADY(By, i) + GRADZ(Bz, i);

                DivB[i] = divB;

                fx = cby * bz - cbz * by;
                fy = cbz * bx - cbx * bz;
                fz = cbx * by - cby * bx;

                oxa[i] = (1.0 / b2) * (fx);
                oya[i] = (1.0 / b2) * (fy);
                oza[i] = (1.0 / b2) * (fz);
                oxb[i] = (1.0 / b2) * (divB * bx);
                oyb[i] = (1.0 / b2) * (divB * by);
                ozb[i] = (1.0 / b2) * (divB * bz);

                o2a = oxa[i] * oxa[i] + oya[i] * oya[i] + oza[i] * oza[i];
                o2b = oxb[i] * oxb[i] + oyb[i] * oyb[i] + ozb[i] * ozb[i];
                helpL = helpL + b2 * o2a + b2 * o2b;
                helpL1 = helpL1 + b2 * o2a;
                helpL2 = helpL2 + b2 * o2b;

                oxbx[i] = oya[i] * bz - oza[i] * by;
                oxby[i] = oza[i] * bx - oxa[i] * bz;
                oxbz[i] = oxa[i] * by - oya[i] * bx;
                odotb[i] = oxb[i] * bx + oyb[i] * by + ozb[i] * bz;
                oxjx[i] = oya[i] * cbz - oza[i] * cby;
                oxjy[i] = oza[i] * cbx - oxa[i] * cbz;
                oxjz[i] = oxa[i] * cby - oya[i] * cbx;
            }
    helpLs[icube] = helpL;
    helpLs[ncube_xyz + icube] = helpL1;
    helpLs[ncube_xyz * 2 + icube] = helpL2;
}

__global__ void update_d(const int *shape, const int calcb, const int boundary, const double doubled_h, const double oldL, const double L, const double Lx, const double nue,
                         const double *Bxorig, const double *Byorig, const double *Bzorig, const double *mask,
                         float *DivB, float *odotb, float *oxbx, float *oxby, float *oxbz,
                             float *oxjx, float *oxjy, float *oxjz, float *oxa, float *oya, float *oza, float *oxb, float *oyb, float *ozb,
                         double *mues, double *Bx, double *By, double *Bz, double *Bx1, double *By1, double *Bz1)
{
    int nx = shape[0], ny = shape[1], nz = shape[2], cube_x = shape[3], cube_y = shape[4], cube_z = shape[5];
    double mue = mues[0], mue2 = mues[1];

    int i, i1, ix, iy, iz;
    int nynz = ny * nz;
    int delta_x = threadIdx.x * cube_x;
    int delta_y = threadIdx.y * cube_y;
    int delta_z = blockIdx.z * cube_z;
    int max_x = (delta_x + cube_x);
    int max_y = (delta_y + cube_y);
    int max_z = (delta_z + cube_z);
    max_x = max_x < nx ? max_x : nx;
    max_y = max_y < ny ? max_y : ny;
    max_z = max_z < nz ? max_z : nz;

    float Fx,Fy,Fz;
    double o2a, o2b;
    double term1x, term2x, term3x, term4x, term5ax, term5bx;
    double term1y, term2y, term3y, term4y, term5ay, term5by;
    double term1z, term2z, term3z, term4z, term5az, term5bz;

    /*** L minimieren ***/
    for (ix = delta_x == 0 ? 1 : delta_x; ix < (max_x == nx ? nx - 1 : max_x); ix++)
        for (iy = delta_y == 0 ? 1 : delta_y; iy < (max_y == ny ? ny - 1 : max_y); iy++)
            for (iz = delta_z == 0 ? 1 : delta_z; iz < (max_z == nz ? nz - 1 : max_z); iz++)
            {
                i = ix * nynz + iy * nz + iz;
                term1x = GRADY(oxbz, i) - GRADZ(oxby, i);
                term1y = GRADZ(oxbx, i) - GRADX(oxbz, i);
                term1z = GRADX(oxby, i) - GRADY(oxbx, i);

                term2x = oxjx[i];
                term2y = oxjy[i];
                term2z = oxjz[i];

                term3x = GRADX(odotb, i);
                term3y = GRADY(odotb, i);
                term3z = GRADZ(odotb, i);

                term4x = oxb[i] * DivB[i];
                term4y = oyb[i] * DivB[i];
                term4z = ozb[i] * DivB[i];

                o2a = oxa[i] * oxa[i] + oya[i] * oya[i] + oza[i] * oza[i];
                o2b = oxb[i] * oxb[i] + oyb[i] * oyb[i] + ozb[i] * ozb[i];
                term5ax = Bx[i] * o2a;
                term5ay = By[i] * o2a;
                term5az = Bz[i] * o2a;
                term5bx = Bx[i] * o2b;
                term5by = By[i] * o2b;
                term5bz = Bz[i] * o2b;

                Fx = (term1x - term2x + term5ax) + (term3x - term4x + term5bx);
                Fy = (term1y - term2y + term5ay) + (term3y - term4y + term5by);
                Fz = (term1z - term2z + term5az) + (term3z - term4z + term5bz);

                Bx1[i] = Bx[i] + 1.0 * mue * Fx;
                By1[i] = By[i] + 1.0 * mue * Fy;
                Bz1[i] = Bz[i] + 1.0 * mue * Fz;
            }
    /*  Try to repair errors in Magnetogram */
    if (calcb == 40)
    {
        iz = 0;
        mue2 = mue;

        for (ix = delta_x; ix < max_x; ix++)
            for (iy = delta_y; iy < max_y; iy++)
            {
                i = ix * nynz + iy * nz + iz;
                i1 = ix * ny + iy;
                Bx1[i] = Bx[i] - 1.0 * mue2 * oxby[i] - nue * mask[i1] * (Bx[i] - Bxorig[i1]);
                By1[i] = By[i] + 1.0 * mue2 * oxbx[i] - nue * mask[i1] * (By[i] - Byorig[i1]);
                Bz1[i] = Bz[i] - nue * (Bz[i] - Bzorig[i1]);
            }
    }

    /* Use G to minmize L at the boundary */
    if ((calcb == 40) && (boundary == 1))
    {
        mue2 = mue / (1.0 * Lx) / 50.0;
        mue2 = mue;
        /* printf("\n Boundary-Relax "); */
        for (ix = delta_x; ix < max_x; ix++)
            for (iy = delta_y; iy < max_y; iy++)
            {
                iz = 0; /* bottom */
                i = ix * nynz + iy * nz + iz;
                iz = nz - 1; /*top */
                i = ix * nynz + iy * nz + iz;
                Bx1[i] = Bx[i] + mue2 * oxby[i];
                By1[i] = By[i] - mue2 * oxbx[i];
                Bz1[i] = 1.0 * (Bz[i] + mue2 * odotb[i]);
            }
        for (ix = delta_x; ix < max_x; ix++)
            for (iz = delta_z; iz < max_z; iz++)
            {
                iy = 0; /* front */
                i = ix * nynz + iy * nz + iz;
                Bx1[i] = Bx[i] + mue2 * oxbz[i];
                By1[i] = 1.0 * (By[i] - mue2 * odotb[i]);
                Bz1[i] = Bz[i] - mue2 * oxbx[i];
                iy = ny - 1; /* back */
                i = ix * nynz + iy * nz + iz;
                Bx1[i] = Bx[i] - mue2 * oxbz[i];
                By1[i] = 1.0 * (By[i] + mue2 * odotb[i]);
                Bz1[i] = Bz[i] + mue2 * oxbx[i];
            }
        for (iy = delta_y; iy < max_y; iy++)
            for (iz = delta_z; iz < max_z; iz++)
            {
                ix = 0; /* left */
                i = ix * nynz + iy * nz + iz;
                Bx1[i] = 1.0 * (Bx[i] - mue2 * odotb[i]);
                By1[i] = By[i] - mue2 * oxbz[i];
                Bz1[i] = Bz[i] + mue2 * oxby[i];
                ix = nx - 1; /* right */
                i = ix * nynz + iy * nz + iz;
                Bx1[i] = 1.0 * (Bx[i] + mue2 * odotb[i]);
                By1[i] = By[i] + mue2 * oxbz[i];
                Bz1[i] = Bz[i] - mue2 * oxby[i];
            }
    }
}

void calculate(const int maxit, const int diagstep, const int calcb, const int boundary, const int nx, const int ny, const int nz, const double dx, const double dy, const double dz, const double Lx, const double nave, const double nue,
               const double *Bxorig, const double *Byorig, const double *Bzorig, const double *mask,
               int &it, double &mue, FILE *streamw, double *Bx, double *By, double *Bz)
{
    int cube_x, cube_y, cube_z, ncube_x, ncube_y, ncube_z;
    cube_x = (nx + 32 - 1) / 32;
    cube_y = (ny + 32 - 1) / 32;
    cube_z = (nz + 32 - 1) / 32;
    cube_x = cube_x < 4 ? 4 : cube_x;
    cube_y = cube_y < 4 ? 4 : cube_y;
    cube_z = cube_z < 4 ? 4 : cube_z;
    ncube_x = (nx + cube_x - 1) / cube_x;
    ncube_y = (ny + cube_y - 1) / cube_y;
    ncube_z = (nz + cube_z - 1) / cube_z;

    int shape[9] = {nx, ny, nz, cube_x, cube_y, cube_z, ncube_x, ncube_y, ncube_z};
    int nxny = nx * ny, nxnynz = nx * ny * nz, statcount = 0, restore = 0;
    int ncube_xyz = ncube_x * ncube_y * ncube_z;
    double mue2, doubled_h = dx * 2, L1 = 0.0, L2 = 0.0, L3 = 0.0, L = 0.0, oldL = 0.0, prevL = 0.0, newL = 0.0, gradL = 0.0;
    double mues[2];

    dim3 blockSize(ncube_x, ncube_y, 1);
    dim3 gridSize(1, 1, ncube_z);

    int *shape_d;
    float *DivB_d, *odotb_d, *oxbx_d, *oxby_d, *oxbz_d, *oxjx_d, *oxjy_d, *oxjz_d, *oxa_d, *oya_d, *oza_d, *oxb_d, *oyb_d, *ozb_d;
    double *mues_d, *Bxorig_d, *Byorig_d, *Bzorig_d, *help_d, *Bx_d, *By_d, *Bz_d, *Bx1_d, *By1_d, *Bz1_d, *Bx_swap_d, *By_swap_d, *Bz_swap_d, *mask_d;

    hipMalloc((void **)&shape_d, 9 * sizeof(int));
    hipMalloc((void **)&DivB_d, nxnynz * sizeof(float));
    hipMalloc((void **)&odotb_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxbx_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxby_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxbz_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxjx_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxjy_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxjz_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxa_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oya_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oza_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oxb_d, nxnynz * sizeof(float));
    hipMalloc((void **)&oyb_d, nxnynz * sizeof(float));
    hipMalloc((void **)&ozb_d, nxnynz * sizeof(float));
    hipMalloc((void **)&mues_d, 2 * sizeof(double));
    hipMalloc((void **)&help_d, 3 * ncube_xyz * sizeof(double));
    hipMalloc((void **)&Bx_d, nxnynz * sizeof(double));
    hipMalloc((void **)&By_d, nxnynz * sizeof(double));
    hipMalloc((void **)&Bz_d, nxnynz * sizeof(double));
    hipMalloc((void **)&Bx1_d, nxnynz * sizeof(double));
    hipMalloc((void **)&By1_d, nxnynz * sizeof(double));
    hipMalloc((void **)&Bz1_d, nxnynz * sizeof(double));
    hipMalloc((void **)&Bxorig_d, nxny * sizeof(double));
    hipMalloc((void **)&Byorig_d, nxny * sizeof(double));
    hipMalloc((void **)&Bzorig_d, nxny * sizeof(double));
    hipMalloc((void **)&mask_d, nxny * sizeof(double));

    hipMemset((float *)DivB_d, (float)0.0, nxnynz * sizeof(float));
    hipMemcpy((void *)shape_d, (void *)shape, 9 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void *)Bxorig_d, (void *)Bxorig, nxny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Byorig_d, (void *)Byorig, nxny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Bzorig_d, (void *)Bzorig, nxny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)mask_d, (void *)mask, nxny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Bx_d, (void *)Bx, nxnynz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)By_d, (void *)By, nxnynz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Bz_d, (void *)Bz, nxnynz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)Bx1_d, (void *)Bx_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy((void *)By1_d, (void *)By_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy((void *)Bz1_d, (void *)Bz_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);

    time1_calculate = clock();
    hipError_t err = hipGetLastError();
    while (it < maxit && statcount < 10 && mue > 1.0e-7 * dx * dx)
    {
        it = it + 1;

        time1_calculate1 = clock();
        calculateL_d<<<gridSize, blockSize>>>(shape_d, doubled_h, Bx_d, By_d, Bz_d, DivB_d, odotb_d, oxbx_d, oxby_d, oxbz_d, oxjx_d, oxjy_d, oxjz_d, oxa_d, oya_d, oza_d, oxb_d, oyb_d, ozb_d, help_d);
        if (err != hipSuccess)
        {
            printf("[calculateL_1_d]: CUDA Error: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        time2_calculate1 = clock();
        timeTot_calculate1 += ((time2_calculate1 - time1_calculate1) / CLOCKS_PER_SEC);

        time1_calculate2 = clock();
        L = sum1D(ncube_xyz, help_d);
        L1 = sum1D(ncube_xyz, help_d + ncube_xyz);
        L2 = sum1D(ncube_xyz, help_d + ncube_xyz * 2);
        L = L * dx * dy * dz;
        L1 = L1 * dx * dy * dz;
        L2 = L2 * dx * dy * dz;
        time2_calculate2 = clock();
        timeTot_calculate2 += ((time2_calculate2 - time1_calculate2) / CLOCKS_PER_SEC);

        if (calcb == 40)
        {
            time1_calculate3 = clock();

            L3 = calculateL3(nx, ny, nz, nave, help_d, mask_d, Bx_d, By_d, Bz_d, Bxorig_d, Byorig_d, Bzorig_d);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("[calculateL3]: CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipDeviceSynchronize();
            L = L + L3;

            time2_calculate3 = clock();
            timeTot_calculate3 += ((time2_calculate3 - time1_calculate3) / CLOCKS_PER_SEC);
        }

        if (it == 0)
            oldL = L;

        if (restore == 1)
            L = oldL; /* restore when L is rejected */

        if (it > 0 && L > oldL) /* Changed L>=oldL to L>oldL */
        {
            restore = 1; /* restore in the next loop step */
            mue = mue / 2.0;
            printf("\n mue reduced, mue= %lf \t mue/dx^2= %lf", mue, mue / (dx * dx));
            printf("\n oldL= %lf \t L=%lf", oldL, L);
            it = it - 1;

            hipMemcpy((void *)Bx_d, (void *)Bx1_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);
            hipMemcpy((void *)By_d, (void *)By1_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);
            hipMemcpy((void *)Bz_d, (void *)Bz1_d, nxnynz * sizeof(double), hipMemcpyDeviceToDevice);
        }
        else
        {
            mue = mue * 1.01;
            restore = 0; /* no restore */
            oldL = L;
        }

        if (oldL >= L)
        {
            mues[0] = mue;
            mues[1] = mue2;
           
            time1_calculate4 = clock();
            
            hipMemcpy((void *)mues_d, (void *)mues, 2 * sizeof(double), hipMemcpyHostToDevice);
            update_d<<<gridSize, blockSize>>>(shape_d, calcb, boundary, doubled_h, oldL, L, Lx, nue, Bxorig_d, Byorig_d, Bzorig_d, mask_d, DivB_d, odotb_d, oxbx_d, oxby_d, oxbz_d, oxjx_d, oxjy_d, oxjz_d, oxa_d, oya_d, oza_d, oxb_d, oyb_d, ozb_d, mues_d, Bx_d, By_d, Bz_d, Bx1_d, By1_d, Bz1_d);
            hipMemcpy((void *)mues, (void *)mues_d, 2 * sizeof(double), hipMemcpyDeviceToHost);
            
            time2_calculate4 = clock();
            timeTot_calculate4 += ((time2_calculate4 - time1_calculate4) / CLOCKS_PER_SEC);
           
            Bx_swap_d = Bx_d;
            By_swap_d = By_d;
            Bz_swap_d = Bz_d;
            Bx_d = Bx1_d;
            By_d = By1_d;
            Bz_d = Bz1_d;
            Bx1_d = Bx_swap_d;
            By1_d = By_swap_d;
            Bz1_d = Bz_swap_d;
            mue = mues[0];
            mue2 = mues[1];
        }

        /*** Some tests with (Anti) Symmetric Boundary-conditions ***/
        if (it % diagstep == 0) /* && L<=oldL ) */
        {
            printf("\n %i L= %.4f", it, L);
            /* NEW: calc gradient (dL/dt)L for stopping rule */
            if (it == 0)
            {
                prevL = 2.0 * L;
                newL = L;
            }
            else
            {
                prevL = newL;
                newL = L;
            }
            gradL = fabs((newL - prevL) / newL);
            if (gradL < 0.0001)
            {
                statcount = statcount + 1;
                printf("\n *** STATIONARY STATE count: %i *** grad L/L= %lf \n", statcount, gradL);
            }
            if (gradL > 0.0001)
            {
                statcount = 0;
            }

            if (calcb != 40)
                printf(",  gradL/L= %lf", gradL);
            if (calcb == 40)
            {
                printf(",L1=%.4lf, L2=%.4lf,  L3= %.4lf", L1, L2, L3);
            }
            if ((streamw = fopen("step.log", "w")) == NULL)
            {
                printf("\n Error help.log");
                exit(1);
            }
            fprintf(streamw, "%i", it / diagstep);
            fclose(streamw);
            if ((streamw = fopen("prot.log", "a")) == NULL)
            {
                printf("\n Error prot.log");
                exit(1);
            }
            fprintf(streamw, "%lf \n %lf \n %lf \n %lf \n %lf \n %lf \n", 1.0 * it, L, gradL, L1, L2, L3);
            fclose(streamw);
        }
        /* End Diagnostik */
    }

    hipMemcpy((void *)Bx, (void *)Bx_d, nxnynz * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy((void *)By, (void *)By_d, nxnynz * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy((void *)Bz, (void *)Bz_d, nxnynz * sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    time2_calculate = clock();
    timeTot_calculate = (time2_calculate - time1_calculate) / CLOCKS_PER_SEC;
    // free();
    // cudaFree();
}
